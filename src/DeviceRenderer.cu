#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2014 Christos Papaioannou
 *
 * This software is provided 'as-is', without any express or implied
 * warranty. In no event will the authors be held liable for any damages
 * arising from the use of this software.
 *
 * Permission is granted to anyone to use this software for any purpose,
 * including commercial applications, and to alter it and redistribute it
 * freely, subject to the following restrictions:
 *
 * 1. The origin of this software must not be misrepresented; you must not
 * claim that you wrote the original software. If you use this software
 * in a product, an acknowledgment in the product documentation would be
 * appreciated but is not required.
 *
 * 2. Altered source versions must be plainly marked as such, and must not be
 * misrepresented as being the original software.
 *
 * 3. This notice may not be removed or altered from any source distribution.
 */

#include "DeviceRenderer.h"
#include "kernelWrapper.h"
#include <cuda_gl_interop.h>

HOST void DeviceRenderer::renderToGLPixelBuffer(GLuint pbo)const {

	/* 
	 * 1. bind opengl resources (just the pbo for now)
	 * 2. invoke cuda kernel and rendering 
	 * 3. unbing gl resources
	 * 4. return
	 */

	void* d_pbo = NULL;
	size_t d_pboSize;
	hipGraphicsResource_t cudaResourcePBO;
	
	// bind
	cudaErrorCheck( hipGraphicsGLRegisterBuffer(&cudaResourcePBO, pbo, hipGraphicsRegisterFlagsWriteDiscard));	// only write
	cudaErrorCheck( hipGraphicsMapResources ( 1, &cudaResourcePBO, 0));
	// get pointer
	cudaErrorCheck( hipGraphicsResourceGetMappedPointer(&d_pbo, &d_pboSize, cudaResourcePBO)); 

	// render
	this->renderToCudaBuffer(d_pbo, d_pboSize);
	// wait for kernel
	cudaErrorCheck( hipDeviceSynchronize());
	// unbind
	cudaErrorCheck( hipGraphicsUnmapResources( 1, &cudaResourcePBO, 0));
	return;
}

HOST void DeviceRenderer::renderSceneToGLPixelBuffer(DScene* h_Dscene, DRayIntersection* intersectionBuffer, int bufferSize, GLuint pbo) const{
	/*
	Camera* d_camera;
	DRayTracer* d_tracer;
	void* d_pbo = NULL;
	size_t d_pboSize;
	hipGraphicsResource_t cudaResourcePBO;
	
	int blockdim[2];
	int threadPerBlock[2];


	// bind
	cudaErrorCheck( hipGraphicsGLRegisterBuffer(&cudaResourcePBO, pbo, hipGraphicsRegisterFlagsWriteDiscard));	// only write
	cudaErrorCheck( hipGraphicsMapResources ( 1, &cudaResourcePBO, 0));
	// get pointer
	cudaErrorCheck( hipGraphicsResourceGetMappedPointer(&d_pbo, &d_pboSize, cudaResourcePBO));

	int width = this->getWidth();
	int height = this->getHeight();

	// (16,16) threads per block
	threadPerBlock[0] = 16;
	threadPerBlock[1] = 16;

	blockdim[0] = width / threadPerBlock[0];
	blockdim[1] = height / threadPerBlock[1];

	d_camera = this->getDeviceCamera();
	d_tracer = this->getDeviceRayTracer();

	cudaErrorCheck( hipMemset(intersectionBuffer, 0, bufferSize * sizeof(DRayIntersection)));
	// invoke intersectio kernel.Traverse the BVH first
	calculateIntersections(d_camera, intersectionBuffer, bufferSize, h_Dscene->m_Triangles, h_Dscene->m_NumTriangles,
						   h_Dscene->m_BvhBuffer, width, height, blockdim, threadPerBlock);
	// wait for kernel
	cudaErrorCheck( hipDeviceSynchronize());

	// shadeIntersections
	shadeIntersectionsToBuffer((uchar4*)d_pbo, d_pboSize, d_tracer, d_camera, h_Dscene->m_Lights, h_Dscene->m_NumLights, intersectionBuffer, bufferSize, 
										h_Dscene->m_Materials, h_Dscene->m_NumMaterials, width, height, blockdim, threadPerBlock);

	cudaErrorCheck( hipDeviceSynchronize());
	cudaErrorCheck( hipGraphicsUnmapResources( 1, &cudaResourcePBO, 0));
	*/
}

HOST void DeviceRenderer::renderToCudaBuffer(void* d_buffer, unsigned int buffer_len)const{
	Camera* d_camera;
	DScene* d_scene;
	DRayTracer* d_tracer;

	int blockdim[2];
	int threadPerBlock[2];

	int width = this->getWidth();
	int height = this->getHeight();

	// (16,16) threads per block
	threadPerBlock[0] = 16;
	threadPerBlock[1] = 16;

	blockdim[0] = width / threadPerBlock[0];
	blockdim[1] = height / threadPerBlock[1];

	d_camera = this->getDeviceCamera();
	d_scene  = this->getDeviceScene();
	d_tracer = this->getDeviceRayTracer();

	renderToBuffer((char*)d_buffer, buffer_len, d_camera, d_scene, d_tracer, blockdim, threadPerBlock, width, height);
}


HOST void  DeviceRenderer::renderToHostBuffer(void* h_buffer, unsigned int buffer_len)const{

	void* d_buffer;
	cudaErrorCheck( hipMalloc(&d_buffer, buffer_len));
	cudaErrorCheck( hipMemset(d_buffer, 0, buffer_len));

	// call kernel
	this->renderToCudaBuffer(d_buffer, buffer_len);

	// wait for computation
	cudaErrorCheck( hipDeviceSynchronize());

	// copy result to host buffer
	cudaErrorCheck( hipMemcpy( h_buffer, d_buffer, buffer_len, hipMemcpyDeviceToHost));
	cudaErrorCheck( hipFree(d_buffer));
}


HOST void DeviceRenderer::renderSceneToHostBuffer(DScene* h_Dscene, cudaIntersection_t* intersectionBuffer, int bufferSize, void* imageBuffer, int imageBufferSize){
	Camera* d_camera;
	DRayTracer* d_tracer;
	void* d_image;
	cudaErrorCheck( hipMalloc(&d_image, imageBufferSize));
	cudaErrorCheck( hipMemset(d_image, 0, imageBufferSize));

	int blockdim[2];
	int threadPerBlock[2];

	int width = this->getWidth();
	int height = this->getHeight();

	threadPerBlock[0] = 16;
	threadPerBlock[1] = 16;

	blockdim[0] = width / threadPerBlock[0];
	blockdim[1] = height / threadPerBlock[1];

	d_camera = this->getDeviceCamera();

	// reset intersections

	//cudaErrorCheck( hipMemset(intersectionBuffer, 0, bufferSize * sizeof(DRayIntersection)));
	cudaErrorCheck( hipMemset(m_CudaHostIntersection->points, 0, sizeof(glm::vec4) * width * height));
	cudaErrorCheck( hipMemset(m_CudaHostIntersection->normals, 0, sizeof(glm::vec4) * width * height));
	cudaErrorCheck( hipMemset(m_CudaHostIntersection->materialsIndices, 0, sizeof(int) * width * height));

	calculateIntersections(d_camera, m_CudaDeviceIntersection, bufferSize, h_Dscene->m_Triangles, h_Dscene->m_NumTriangles,
						   h_Dscene->m_BvhBuffer, width, height, blockdim, threadPerBlock);

	d_tracer = this->getDeviceRayTracer();

	cudaErrorCheck( hipDeviceSynchronize());

		// shadeIntersections
	shadeIntersectionsToBuffer((uchar4*)d_image, imageBufferSize, d_tracer, d_camera, h_Dscene->m_Lights, h_Dscene->m_NumLights, m_CudaDeviceIntersection, bufferSize, 
										h_Dscene->m_Materials, h_Dscene->m_NumMaterials, width, height, blockdim, threadPerBlock);

	cudaErrorCheck( hipDeviceSynchronize());

	cudaErrorCheck( hipMemcpy( imageBuffer, d_image, imageBufferSize, hipMemcpyDeviceToHost));
	cudaErrorCheck( hipFree(d_image));

}
HOST void DeviceRenderer::setCamera(Camera* d_camera){
	m_Camera = d_camera;
}

HOST void DeviceRenderer::allocateCudaIntersectionBuffer(){
	// allocate cudaIntersection		
	glm::vec4* d_pointsVec4;
	glm::vec4* d_normalsVec4;
	int* d_materialIndices;
	int width;
	int height;

	width = this->getWidth();
	height = this->getHeight();

	cudaErrorCheck( hipMalloc((void**) &d_pointsVec4, sizeof(glm::vec4) * width * height));
	cudaErrorCheck( hipMalloc((void**) &d_normalsVec4, sizeof(glm::vec4) * width * height));
	cudaErrorCheck( hipMalloc((void**) &d_materialIndices, sizeof(int) * width * height));

	cudaErrorCheck( hipMemset(d_pointsVec4, 0, sizeof(glm::vec4) * width * height));
	cudaErrorCheck( hipMemset(d_normalsVec4, 0, sizeof(glm::vec4) * width * height));
	cudaErrorCheck( hipMemset(d_materialIndices, 0, sizeof(int) * width * height));

	
	cudaIntersection_t* deviceIntersection;

	m_CudaHostIntersection = new cudaIntersection_t;
	m_CudaHostIntersection->points  = d_pointsVec4 ;
	m_CudaHostIntersection->normals = d_normalsVec4;
	m_CudaHostIntersection->materialsIndices = d_materialIndices;

	cudaErrorCheck( hipMalloc((void**) &deviceIntersection, sizeof(cudaIntersection_t)));
	cudaErrorCheck( hipMemcpy( deviceIntersection, m_CudaHostIntersection, sizeof(cudaIntersection_t), hipMemcpyHostToDevice));

	m_CudaDeviceIntersection = deviceIntersection;
}