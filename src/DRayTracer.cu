#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2014 Christos Papaioannou
 *
 * This software is provided 'as-is', without any express or implied
 * warranty. In no event will the authors be held liable for any damages
 * arising from the use of this software.
 *
 * Permission is granted to anyone to use this software for any purpose,
 * including commercial applications, and to alter it and redistribute it
 * freely, subject to the following restrictions:
 *
 * 1. The origin of this software must not be misrepresented; you must not
 * claim that you wrote the original software. If you use this software
 * in a product, an acknowledgment in the product documentation would be
 * appreciated but is not required.
 *
 * 2. Altered source versions must be plainly marked as such, and must not be
 * misrepresented as being the original software.
 *
 * 3. This notice may not be removed or altered from any source distribution.
 */

#include "DRayTracer.h"


DEVICE glm::vec4 DRayTracer::rayTrace(DScene* scene, Camera* camera, const Ray& ray,  int depth){
	DRayIntersection intersection;
	
	if( depth > this->getTracedDepth() )
		return glm::vec4(0.0f);
	// find itersection
	if( scene->isUsingBVH() ){
		if( scene->findMinDistanceIntersectionBVH(ray, intersection))
			return this->shadeIntersection(scene, ray, camera, intersection, depth);

	}
	else{
		if( scene->findMinDistanceIntersectionLinear( ray, intersection)){

			return this->shadeIntersection(scene, ray, camera, intersection, depth);
		}
	}
	return glm::vec4(0.0f);
}

DEVICE glm::vec4 DRayTracer::shadeIntersection(DScene* scene, const Ray& ray, Camera* camera, DRayIntersection& intersection, int depth){
	int numLights;
	int i;
	glm::vec4 finalColor(0.0f);

	numLights = scene->getNumLights();
	for( i = 0; i < numLights; i++){
		finalColor += this->calcPhong(camera, scene->getLightSource(i), intersection);
	}
	return finalColor;
}

DEVICE glm::vec4 DRayTracer::calcPhong(Camera* camera, DLightSource* lightSource, DRayIntersection& intersection){
	glm::vec4 color(0.0f, 0.0f, 0.0f, 0.0f);
	//glm::vec4 specularColor(0.0f, 0.0f, 0.0f, 0.0f);
	//glm::vec4 intersectionToLight;
	//glm::vec4 viewVector;
	//glm::vec4 reflectedVector;

	//glm::vec4 intersectionPointInWorld  = glm::vec4(intersection.getIntersectionPoint() , 1.0f);
	//glm::vec4 intersectionNormalInWorld = glm::vec4(intersection.getIntersectionNormal(), 0.0f);

	
	// specular reflection
	glm::vec4 intersectionToLight = glm::normalize(lightSource->getPosition() - glm::vec4(intersection.getIntersectionPoint() , 1.0f));
	//viewVector          = glm::normalize(glm::vec4(camera->getPosition(),1.0f) - glm::vec4(intersection.getIntersectionPoint() , 1.0f));
	//reflectedVector     = glm::normalize((2.0f * glm::dot(glm::vec4(intersection.getIntersectionNormal(), 0.0f), intersectionToLight) * glm::vec4(intersection.getIntersectionNormal(), 0.0f)) - intersectionToLight);
	
	// find diffuse first
	//diffuseColor = this->findDiffuseColor(lightSource, glm::normalize(lightSource->getPosition() - glm::vec4(intersection.getIntersectionPoint() , 1.0f)), intersection);

	float dot = glm::dot( glm::normalize(glm::vec4(camera->getPosition(),1.0f) - glm::vec4(intersection.getIntersectionPoint() , 1.0f)), glm::normalize((2.0f * glm::dot(glm::vec4(intersection.getIntersectionNormal(), 0.0f), intersectionToLight) * glm::vec4(intersection.getIntersectionNormal(), 0.0f)) - intersectionToLight));
	if( dot > 0.0f){
		float specularTerm = glm::pow(dot, (float)intersection.getIntersectionMaterial().getShininess());
		color += specularTerm * lightSource->getColor() * intersection.getIntersectionMaterial().getSpecularColor();
	}

	return this->findDiffuseColor(lightSource, glm::normalize(lightSource->getPosition() - glm::vec4(intersection.getIntersectionPoint() , 1.0f)), intersection) + color;

}

DEVICE glm::vec4 DRayTracer::findDiffuseColor(DLightSource* lightSource, const glm::vec4& intersectionToLight, DRayIntersection& intersection){
	
	const DMaterial& material = intersection.getIntersectionMaterial();
	float dot = glm::dot(intersectionToLight, glm::vec4(intersection.getIntersectionNormal(), 0.0f));
	dot = glm::max(0.0f, dot);
	return glm::vec4( dot * material.getDiffuseColor() * lightSource->getColor());
}