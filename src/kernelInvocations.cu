#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2014 Christos Papaioannou
 *
 * This software is provided 'as-is', without any express or implied
 * warranty. In no event will the authors be held liable for any damages
 * arising from the use of this software.
 *
 * Permission is granted to anyone to use this software for any purpose,
 * including commercial applications, and to alter it and redistribute it
 * freely, subject to the following restrictions:
 *
 * 1. The origin of this software must not be misrepresented; you must not
 * claim that you wrote the original software. If you use this software
 * in a product, an acknowledgment in the product documentation would be
 * appreciated but is not required.
 *
 * 2. Altered source versions must be plainly marked as such, and must not be
 * misrepresented as being the original software.
 *
 * 3. This notice may not be removed or altered from any source distribution.
 */

#include "kernelInvocations.h"
#include "Ray.h"

/* ===============  KERNELS =================*/
/*
 * Simple kernel
 * one thread/pixel
 */
__global__ void __oneThreadPerPixel_kernel(){


}

__global__ void __renderToBuffer_kernel(char* buffer, unsigned int buffer_len, Camera* camera, DScene* scene, DRayTracer* rayTracer, int width, int height){
	
	
    int pi = blockIdx.x * blockDim.x + threadIdx.x;
    int pj = blockIdx.y * blockDim.y + threadIdx.y;
     

    if (pi < width && pj < height){

    	// generate ray
    	Ray ray;
  
    	//float bb = (pj - norm_height) / norm_height;
    	//float aa = (pi - norm_width) / norm_width;
        //float aa = ((2.0f * pi - width) / (float) width);
    	//float bb = ((2.0f * pj - height) / (float) height);

    	ray.setOrigin(camera->getPosition());
    	ray.setDirection( glm::normalize((((2.0f * pi - width) / (float) width) * camera->getRightVector() ) + ( ((2.0f * pj - height) / (float) height) * camera->getUpVector()) + camera->getViewingDirection()));
 		
 		// find color
 		glm::vec4 color = rayTracer->rayTrace(scene, camera, ray, 0);	// depth = 0

 		// store color
    	buffer[4 * (pi + pj * width)]      = floor(color.x == 1.0 ? 255 : fminf(color.x * 256.0f, 255.0f));
        buffer[1 +  4* (pi + pj * width)]  = floor(color.y == 1.0 ? 255 : fminf(color.y * 256.0f, 255.0f));
        buffer[2 +  4* (pi + pj * width)]  = floor(color.z == 1.0 ? 255 : fminf(color.z * 256.0f, 255.0f));
        buffer[3 +  4* (pi + pj * width)]  = (char)255;
    }     

}


/* ============ WRAPPERS ====================*/

void renderToBuffer(char* buffer, unsigned int buffer_len, Camera* camera, DScene* scene, DRayTracer* rayTracer, int blockdim[], int tpblock[], int width, int height){

	dim3 threadsPerBlock;
	dim3 numBlocks;

	threadsPerBlock.x = tpblock[0];
	threadsPerBlock.y = tpblock[1];

	numBlocks.x = blockdim[0];
	numBlocks.y = blockdim[1];

	__renderToBuffer_kernel<<<numBlocks, threadsPerBlock>>>(buffer, buffer_len, camera, scene, rayTracer, width, height);
}