/*
 * Copyright (c) 2014 Christos Papaioannou
 *
 * This software is provided 'as-is', without any express or implied
 * warranty. In no event will the authors be held liable for any damages
 * arising from the use of this software.
 *
 * Permission is granted to anyone to use this software for any purpose,
 * including commercial applications, and to alter it and redistribute it
 * freely, subject to the following restrictions:
 *
 * 1. The origin of this software must not be misrepresented; you must not
 * claim that you wrote the original software. If you use this software
 * in a product, an acknowledgment in the product documentation would be
 * appreciated but is not required.
 *
 * 2. Altered source versions must be plainly marked as such, and must not be
 * misrepresented as being the original software.
 *
 * 3. This notice may not be removed or altered from any source distribution.
 */

#include "DeviceSceneImporter.h"


/**
 * Copy entire scene from host to device memory
 *
 * BVH is not copied now.
 * returns: pointer to device allocated DScene class
 */

HOST DScene* DeviceSceneImporter::createDeviceScene(){
	
	DScene* d_scene = NULL;
	DScene* h_DScene;
	Scene* h_scene;
	DLightSource* d_LightsArray;
	DTriangle* d_TriangleArray;
	int numLights;
	int numTriangles;

	// create a temp copy of DScene on host
	h_DScene = new DScene;

	cudaErrorCheck( hipMalloc((void**)&d_scene, sizeof(DScene)) );
	if(!d_scene)
		return NULL;

	h_scene = this->getScene();
	numLights = h_scene->getNumLightSources();
	numTriangles = h_scene->getNumSurfaces();

	// set number of triangles and lights 
	h_DScene->m_NumLights = numLights;
	h_DScene->m_NumTriangles = numTriangles;

	// create buffer of DLightSource objects on host
	// and then copy to Device memory
	DLightSource* h_DLightSources = new DLightSource[numLights];
	
	cudaErrorCheck( hipMalloc((void**)&d_LightsArray, sizeof(DLightSource) * numLights));
	if(!d_LightsArray){
		cudaErrorCheck( hipFree(d_scene));
		delete h_DLightSources;
		return NULL;
	}
	
	for( int i = 0; i < numLights; i++){
		const LightSource* h_Light = h_scene->getLightSource(i);
		
		h_DLightSources[i].m_Position = h_Light->getPosition();
		h_DLightSources[i].m_Color = h_Light->getPosition();
	}
	// maybe transfer later ???
	cudaErrorCheck( hipMemcpy(d_LightsArray, h_DLightSources, sizeof(DLightSource) * numLights, hipMemcpyHostToDevice));
	// ATTENTION: d_LightsArray points to GPU memory.DONT DEREFERENCE ON HOST
	h_DScene->m_Lights = d_LightsArray;

	// Now copy triangles to device memory

	DTriangle* h_DTriangles = new DTriangle[numTriangles];
	for(int i = 0 ; i < numTriangles; i++){
		
		Surface* h_Surface = h_scene->getSurface(i);

		Triangle* h_Triangle = dynamic_cast<Triangle*>(h_Surface);
		
		if( !h_Triangle){
			// this surface is not a Triangle.
			// Only scenes containing Triangles will be copied to gpu.
			// cleanup and return null
			cudaErrorCheck( hipFree(d_scene));
			cudaErrorCheck( hipFree(d_LightsArray));
			delete h_DLightSources;
			delete h_DTriangles;
			return NULL;
		}
		//vertices
		h_DTriangles[i].m_V1 = h_Triangle->m_V1;
		h_DTriangles[i].m_V2 = h_Triangle->m_V2;
		h_DTriangles[i].m_V3 = h_Triangle->m_V3;
		// normals
		h_DTriangles[i].m_N1 = h_DTriangles->m_N1;
		h_DTriangles[i].m_N2 = h_DTriangles->m_N2;
		h_DTriangles[i].m_N3 = h_DTriangles->m_N3;
		// copy transformations
		h_DTriangles[i].m_Transformation   = h_Triangle->transformation();
		h_DTriangles[i].m_Inverse          = h_Triangle->getInverseTransformation();
		h_DTriangles[i].m_InverseTranspose = h_Triangle->getInverseTransposeTransformation();
		// material
		DMaterial h_DMaterial;
		const Material& h_TriangleMaterial = h_Triangle->getMaterial(); 
		
		h_DMaterial.m_Diffuse           = h_TriangleMaterial.getDiffuseColor();
		h_DMaterial.m_Specular          = h_TriangleMaterial.getSpecularColor();
		h_DMaterial.m_AmbientIntensity  = h_TriangleMaterial.getAmbientIntensity();
		h_DMaterial.m_Reflectivity      = h_TriangleMaterial.getReflectiveIntensity();
		h_DMaterial.m_shininess         = h_TriangleMaterial.getShininess();

		h_DTriangles[i].m_Material = h_DMaterial;  
	}

	cudaErrorCheck( hipMalloc((void**)&d_TriangleArray, sizeof(DTriangle) * numTriangles));
	cudaErrorCheck( hipMemcpy(d_TriangleArray, h_DTriangles, sizeof(DTriangle) * numTriangles, hipMemcpyHostToDevice));
	h_DScene->m_Triangles = d_TriangleArray;

	// also copy h_DScene
	cudaErrorCheck( hipMemcpy(d_scene, h_DScene, sizeof(DScene), hipMemcpyHostToDevice)); 

	delete h_DScene;
	delete h_DLightSources;
	delete h_DTriangles;
	return d_scene;
}
